#include "hip/hip_runtime.h"
﻿//By Farhang Nemati 2022

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "scope_profile.h"

#include "helper.h"

#define Mask_width 5
#define Mask_radius Mask_width / 2
#define TILE_WIDTH 16
#define TWidth 16
#define w (TILE_WIDTH + Mask_width - 1)
#define clamp(x) (min(max((x), 0.0), 1.0))

//Sequential filtering
void sequential(unsigned char *inputImageData, int *maskData, unsigned char *outputImageData,
                const int imageWidth, const int imageHeight, const int channels, const int maskWidth, int divideBy) {
    int maskRadius = maskWidth / 2;

    for (int i = 0; i < imageHeight; i++) {
        for (int j = 0; j < imageWidth; j++) {
            for (int k = 0; k < channels; k++) {
                float accum = 0;
                for (int y = -maskRadius; y < maskRadius; y++) {
                    for (int x = -maskRadius; x < maskRadius; x++) {
                        unsigned int xOffset = j + x;
                        unsigned int yOffset = i + y;
                        if (xOffset >= 0 && xOffset < imageWidth && yOffset >= 0 && yOffset < imageHeight) {
                            unsigned char imagePixel = inputImageData[(yOffset * imageWidth + xOffset) * channels + k];
                            int maskValue = maskData[(y + maskRadius) * maskWidth + x + maskRadius];
                            accum += imagePixel * maskValue;
                        }
                    }
                }
                // pixels are in the range of 0 to 1
                outputImageData[(i * imageWidth + j) * channels + k] = accum / divideBy;
            }
        }
    }
}

__global__ void convolution(unsigned char *inputImageData, int *maskData, unsigned char *outputImageData,
                            const int imageWidth, const int imageHeight, const int channels, const int maskWidth,
                            int divideBy) {

    __shared__ unsigned char MShared[TWidth + 2 * Mask_radius][TWidth + 2 * Mask_radius];

    auto ySh = threadIdx.y + Mask_radius;
    auto xSh = threadIdx.x + Mask_radius;

    auto col = blockIdx.x * TWidth + threadIdx.x;
    auto row = blockIdx.y * TWidth + threadIdx.y;

    for (int channelIdx = 0; channelIdx < channels; channelIdx++) {
        // the threads in the green and red regions
        if (threadIdx.x == 0 || threadIdx.x == TWidth - 1 || threadIdx.y == 0 || threadIdx.y == TWidth - 1) {
            if (threadIdx.x == 0) {
                auto imageIdx = (row * imageWidth + col) * channels + channelIdx;
                if (imageIdx > 0 && imageIdx < imageHeight * imageWidth * channels)
                    MShared[ySh][xSh] = inputImageData[imageIdx];
                else
                    MShared[ySh][xSh] = 0;

                // 1 pixel behind
                imageIdx = (row * imageWidth + col - 1) * channels + channelIdx;
                if (imageIdx > 0 && imageIdx < imageHeight * imageWidth * channels)
                    MShared[ySh][xSh - 1] = inputImageData[imageIdx];
                else
                    MShared[ySh][xSh - 1] = 0;

                // 2 pixels behind
                imageIdx = (row * imageWidth + col - 2) * channels + channelIdx;

                if (imageIdx > 0 && imageIdx < imageHeight * imageWidth * channels)
                    MShared[ySh][xSh - 2] = inputImageData[imageIdx];
                else
                    MShared[ySh][xSh - 2] = 0;
            }

            if (threadIdx.y == 0) {
                auto imageIdx = (row * imageWidth + col) * channels + channelIdx;
                if (imageIdx > 0 && imageIdx < imageHeight * imageWidth * channels)
                    MShared[ySh][xSh] = inputImageData[imageIdx];
                else
                    MShared[ySh][xSh] = 0;

                // 1 pixel above
                imageIdx = ((row - 1) * imageWidth + col) * channels + channelIdx;
                if (imageIdx > 0 && imageIdx < imageHeight * imageWidth * channels)
                    MShared[ySh - 1][xSh] = inputImageData[imageIdx];
                else
                    MShared[ySh - 1][xSh] = 0;

                // 2 pixels above
                imageIdx = ((row - 2) * imageWidth + col) * channels + channelIdx;
                if (imageIdx > 0 && imageIdx < imageHeight * imageWidth * channels)
                    MShared[ySh - 2][xSh] = inputImageData[imageIdx];
                else
                    MShared[ySh - 2][xSh] = 0;
            }

            // The thread on the corner needs to load 4 more pixels
            if (threadIdx.x == 0 && threadIdx.y == 0) {
                auto imageIdx = ((row - 1) * imageWidth + col - 1) * channels + channelIdx;
                if (imageIdx > 0 && imageIdx < imageHeight * imageWidth * channels)
                    MShared[ySh - 1][xSh - 1] = inputImageData[imageIdx];
                else
                    MShared[ySh - 1][xSh - 1] = 0;

                imageIdx = ((row - 2) * imageWidth + col - 1) * channels + channelIdx;
                if (imageIdx > 0 && imageIdx < imageHeight * imageWidth * channels)
                    MShared[ySh - 2][xSh - 1] = inputImageData[imageIdx];
                else
                    MShared[ySh - 2][xSh - 1] = 0;

                imageIdx = ((row - 1) * imageWidth + col - 2) * channels + channelIdx;
                if (imageIdx > 0 && imageIdx < imageHeight * imageWidth * channels)
                    MShared[ySh - 1][xSh - 2] = inputImageData[imageIdx];
                else
                    MShared[ySh - 1][xSh - 2] = 0;
            }

            auto imageIdx = ((row - 2) * imageWidth + col - 2) * channels + channelIdx;
            if (imageIdx > 0 && imageIdx < imageHeight * imageWidth * channels)
                MShared[ySh - 2][xSh - 2] = inputImageData[imageIdx];
            else
                MShared[ySh - 2][xSh - 2] = 0;

            if (threadIdx.x == TWidth - 1) {
                imageIdx = (row * imageWidth + col) * channels + channelIdx;
                if (imageIdx > 0 && imageIdx < imageHeight * imageWidth * channels)
                    MShared[ySh][xSh] = inputImageData[imageIdx];
                else
                    MShared[ySh][xSh] = 0;

                // 1 pixel after
                imageIdx = (row * imageWidth + col + 1) * channels + channelIdx;
                if (imageIdx > 0 && imageIdx < imageHeight * imageWidth * channels)
                    MShared[ySh][xSh + 1] = inputImageData[imageIdx];
                else
                    MShared[ySh][xSh + 1] = 0;

                // 2 pixels after
                imageIdx = (row * imageWidth + col + 2) * channels + channelIdx;
                if (imageIdx > 0 && imageIdx < imageHeight * imageWidth * channels)
                    MShared[ySh][xSh + 2] = inputImageData[imageIdx];
                else
                    MShared[ySh][xSh + 2] = 0;
            }

            // The other borders will be similar
            // Continue with the same pattern for the rest of border pixels ...

            { // The additional borders ...

                if (threadIdx.y == TWidth - 1) {
                    imageIdx = (row * imageWidth + col) * channels + channelIdx;
                    if (imageIdx > 0 && imageIdx < imageHeight * imageWidth * channels)
                        MShared[ySh][xSh] = inputImageData[imageIdx];
                    else
                        MShared[ySh][xSh] = 0;

                    // 1 pixel after
                    imageIdx = (row * imageWidth + col + 1) * channels + channelIdx;
                    if (imageIdx > 0 && imageIdx < imageHeight * imageWidth * channels)
                        MShared[ySh + 1][xSh] = inputImageData[imageIdx];
                    else
                        MShared[ySh + 1][xSh] = 0;

                    // 2 pixels after
                    imageIdx = (row * imageWidth + col + 2) * channels + channelIdx;
                    if (imageIdx > 0 && imageIdx < imageHeight * imageWidth * channels)
                        MShared[ySh + 2][xSh] = inputImageData[imageIdx];
                    else
                        MShared[ySh + 2][xSh] = 0;
                }

            }

        } else { //the thread in the blue region
            auto imageIdx = (row * imageWidth + col) * channels + channelIdx;
            if (imageIdx > 0 && imageIdx < imageHeight * imageWidth * channels)
                MShared[ySh][xSh] = inputImageData[imageIdx];
            else
                MShared[ySh][xSh] = 0;
        }

        // Wait until all the elements are read by the threads of the block into the shared memory
        __syncthreads();

        // The calculation done by each thread for case MWidth = 5 -> MRadius = 2
        // Replace 2s by MRadius for general case
        auto accum = 0;
        for (int x = -2; x <= 2; ++x) {
            for (int y = -2; y <= 2; ++y) {
                accum += MShared[ySh + x][xSh + y] * maskData[(x + 2) * maskWidth + y + 2];
            }
        }

        auto imageIdx = (row * imageWidth + col) * channels + channelIdx;
        outputImageData[imageIdx] = accum / divideBy;

        // Wait until all the the threads are done with the calculations
        __syncthreads();
    }
}


int maskRows;
int maskColumns;
int imageChannels;
int imageWidth;
int imageHeight;
unsigned char *hostInputImageData;
unsigned char *hostOutputImageData;
int *hostMaskData;
unsigned char *deviceInputImageData;
unsigned char *deviceOutputImageData;
int *deviceMaskData;

//To be divided by 256
int mask1[5][5] = {
        {1, 4,  6,  4,  1},
        {4, 16, 24, 16, 4},
        {6, 24, 36, 24, 6},
        {4, 16, 24, 16, 4},
        {1, 4,  6,  4,  1}
};

//To be divided by 25
int mask2[5][5] = {
        {1, 1, 1, 1, 1},
        {1, 1, 1, 1, 1},
        {1, 1, 1, 1, 1},
        {1, 1, 1, 1, 1},
        {1, 1, 1, 1, 1}
};

int mask3[3][3] = {
        {-1, 0, 1},
        {-2, 0, 2},
        {-1, 0, 1}
};

int mask4[3][3] = {
        {0,  -1, 0},
        {-1, 5,  -1},
        {0,  -1, 0}
};

int main(int argc, char *argv[]) {
    maskRows = 5;
    maskColumns = 5;
    hostMaskData = (int *) malloc(maskRows * maskColumns * sizeof(int));


    if (argc == 2) {
        hostInputImageData = readppm(argv[1], (int *) &imageWidth, (int *) &imageHeight);
    } else if (argc > 2) {
        hostInputImageData = readppm(argv[1], (int *) &imageWidth, (int *) &imageHeight);

        FILE *fp = fopen(argv[2], "rb");
        if (fp == NULL) {
            printf("Mask file could not be found\n");
            return -1;
        }
        fread(hostMaskData, sizeof(int), maskRows * maskColumns, fp);
    } else {
        hostInputImageData = readppm((char *) "lenna512.ppm", (int *) &imageWidth, (int *) &imageHeight);

        for (int i = 0; i < maskRows; i++)
            for (int j = 0; j < maskColumns; j++)
                hostMaskData[i * maskColumns + j] = mask1[i][j];
    }

    imageChannels = 3;
    hostOutputImageData = (unsigned char *) malloc(imageWidth * imageHeight * sizeof(unsigned char) * imageChannels);

    {
        SCOPED_PROFILE_LOG("SEQUENTIAL")

        //Mask1: Gaussian filter
        sequential(hostInputImageData, hostMaskData, hostOutputImageData, imageWidth, imageHeight, imageChannels, 5,
                   256);

        //Mask2
        // sequential(hostInputImageData, hostMaskData, hostOutputImageData, imageWidth, imageHeight, imageChannels, 5, 25);

        //Mask3
        // sequential(hostInputImageData, hostMaskData, hostOutputImageData, imageWidth, imageHeight, imageChannels, 3, 1);

    }

    writeppm("outputImageCpu.ppm", imageWidth, imageHeight, hostOutputImageData);


    {
        SCOPED_PROFILE_LOG("ON GPU")

        const int imageSize = imageWidth * imageHeight * sizeof(unsigned char) * imageChannels;
        const int maskSize = maskRows * maskColumns * sizeof(int);

        printf("Doing GPU memory allocation\n");

        hipError_t err;

        err = hipMalloc(&deviceInputImageData, imageSize);
        if (err != hipSuccess) {
            printf("Failed to allocate device memory for input image data (%s)\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        err = hipMalloc(&deviceMaskData, maskSize);
        if (err != hipSuccess) {
            printf("Failed to allocate device memory for mask data (%s)\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        err = hipMalloc(&deviceOutputImageData, imageSize);
        if (err != hipSuccess) {
            printf("Failed to allocate device memory for output image data (%s)\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        printf("Copying data to the GPU\n");

        err = hipMemcpy(deviceInputImageData, hostInputImageData, imageSize, hipMemcpyHostToDevice);
        if (err != hipSuccess) {
            printf("Failed to copy input image data from host to device (%s)\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        err = hipMemcpy(deviceMaskData, hostMaskData, maskSize, hipMemcpyHostToDevice);
        if (err != hipSuccess) {
            printf("Failed to copy mask data from host to device (%s)\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        printf("Doing the computation on the GPU\n");

        dim3 dBlock(TWidth, TWidth);
        dim3 dGrid(ceil((float) imageWidth / TWidth), ceil((float) imageHeight / TWidth));

        convolution<<<dGrid, dBlock>>>(deviceInputImageData, deviceMaskData, deviceOutputImageData,
                                       imageWidth, imageHeight, imageChannels, 5, 256);

        err = hipGetLastError();
        if (err != hipSuccess) {
            printf("CUDA error: %s\n", hipGetErrorString(err));
        }

        printf("Copying data from the GPU\n");

        err = hipMemcpy(hostOutputImageData, deviceOutputImageData, imageSize, hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
            printf("Failed to copy output image data from device to host (%s)\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

    }


    writeppm("outputImageGpu.ppm", imageWidth, imageHeight, hostOutputImageData);
    return 0;
}